#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <chrono>
#include <fstream>

#define N 1000000  // Размер вектора

__global__ void vectorAdd(const float *A, const float *B, float *C, long n) {
    long idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        C[idx] = A[idx] + B[idx];
    }
}

float runKernel(long threadsPerBlock) {
    long blocksPerGrid = N / threadsPerBlock;

    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, N * sizeof(float));
    hipMalloc((void **)&d_B, N * sizeof(float));
    hipMalloc((void **)&d_C, N * sizeof(float));

    std::vector<float> h_A(N, 1.0f);
    std::vector<float> h_B(N, 2.0f);
    hipMemcpy(d_A, h_A.data(), N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), N * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(h_A.data(), d_C, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return milliseconds;
}

int main() {
    std::ofstream outFile("result.csv");

    for (long T = 16; T <= 512; T += 16) {
        float time = runKernel(T);
        outFile << T << " " << time << "\n";
    }

    outFile.close();

    return 0;
}

