#include <stdio.h>
#include <hip/hip_runtime.h>

#define TILE_DIM 32

__global__ void transposeNaive(float *odata, const float *idata, int width, int height) 
{
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    
    odata[x * height + y] = idata[y * width + x];
}

__global__ void transposeShared(float *odata, const float *idata, int width, int height) 
{
    __shared__ float tile[TILE_DIM][TILE_DIM];

    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    
    tile[threadIdx.y][threadIdx.x] = idata[y * width + x];

    __syncthreads();

    x = blockIdx.y * TILE_DIM + threadIdx.x;
    y = blockIdx.x * TILE_DIM + threadIdx.y;
    
    odata[y * height + x] = tile[threadIdx.x][threadIdx.y];
}

__global__ void transposeSharedNoBankConflicts(float *odata, const float *idata, int width, int height) 
{
    __shared__ float tile[TILE_DIM][TILE_DIM + 1];

    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    
    tile[threadIdx.y][threadIdx.x] = idata[y * width + x];

    __syncthreads();

    x = blockIdx.y * TILE_DIM + threadIdx.x;
    y = blockIdx.x * TILE_DIM + threadIdx.y;
    
    odata[y * height + x] = tile[threadIdx.x][threadIdx.y];
}

int main() 
{
    int width = 1024;
    int height = 1024;
    size_t size = width * height * sizeof(float);

    float *h_in = (float*)malloc(size);
    float *h_out = (float*)malloc(size);

    for (int i = 0; i < width * height; ++i) 
        h_in[i] = (float)i;

    float *d_in, *d_out;
    hipMalloc((void**)&d_in, size);
    hipMalloc((void**)&d_out, size);
    hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);

    dim3 grid((width + TILE_DIM - 1) / TILE_DIM, (height + TILE_DIM - 1) / TILE_DIM);
    dim3 threads(TILE_DIM, TILE_DIM);

    hipEvent_t start, stop;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    #ifndef Native
    hipEventRecord(start, 0);
    transposeNaive<<<grid, threads>>>(d_out, d_in, width, height);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Transpose Naive execution time: %f ms\n", elapsedTime);
    #endif

    #ifndef SharedConflict
    hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);
    hipEventRecord(start, 0);
    transposeShared<<<grid, threads>>>(d_out, d_in, width, height);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Transpose Shared (with conflicts) execution time: %f ms\n", elapsedTime);
    #endif

    #ifndef SharedNoConflict
    hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);
    hipEventRecord(start, 0);
    transposeSharedNoBankConflicts<<<grid, threads>>>(d_out, d_in, width, height);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Transpose Shared (no bank conflicts) execution time: %f ms\n", elapsedTime);
    #endif

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_in);
    hipFree(d_out);
    free(h_in);
    free(h_out);

    return 0;
}
