#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>

#define TILE_DIM 32

__global__ void transposeNaive(float *odata, const float *idata, int width, int height) 
{
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    
    odata[x * height + y] = idata[y * width + x];
}

__global__ void transposeShared(float *odata, const float *idata, int width, int height) 
{
    __shared__ float tile[TILE_DIM][TILE_DIM];

    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    
    tile[threadIdx.y][threadIdx.x] = idata[y * width + x];

    __syncthreads();

    x = blockIdx.y * TILE_DIM + threadIdx.x;
    y = blockIdx.x * TILE_DIM + threadIdx.y;
    
    odata[y * height + x] = tile[threadIdx.x][threadIdx.y];
}

__global__ void transposeSharedNoBankConflicts(float *odata, const float *idata, int width, int height) 
{
    __shared__ float tile[TILE_DIM][TILE_DIM + 1];

    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    
    tile[threadIdx.y][threadIdx.x] = idata[y * width + x];

    __syncthreads();

    x = blockIdx.y * TILE_DIM + threadIdx.x;
    y = blockIdx.x * TILE_DIM + threadIdx.y;
    
    odata[y * height + x] = tile[threadIdx.x][threadIdx.y];
}

void transposeOnCPU(float *odata, const float *idata, int width, int height)
{    
    for(int i = 0; i < height; ++i)
    {
        for(int j = 0; j < width; ++j)
        {
            odata[j * height + i] = idata[i * width + j];
        }
    }
}

int main() 
{
    int width = 1024;
    int height = 1024;

    printf("Size: %d x %d\n", width, height);
    size_t size = width * height * sizeof(float);

    float *h_in = (float*)malloc(size);
    float *h_out = (float*)malloc(size);

    for (int i = 0; i < width * height; ++i) 
        h_in[i] = (float)i;

    float *d_in, *d_out;
    hipMalloc((void**)&d_in, size);
    hipMalloc((void**)&d_out, size);
    hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);

    dim3 grid((width + TILE_DIM - 1) / TILE_DIM, (height + TILE_DIM - 1) / TILE_DIM);
    dim3 threads(TILE_DIM, TILE_DIM);

    hipEvent_t start, stop;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    auto startCpu = std::chrono::high_resolution_clock::now();
    transposeOnCPU(h_out, h_in, width, height);
    auto endCpu = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> duration = endCpu - startCpu;
    printf("Transpose on CPU execution time: %f ms\n", duration.count() * 1000);


    #ifndef Native
    hipEventRecord(start, 0);
    transposeNaive<<<grid, threads>>>(d_out, d_in, width, height);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Transpose Naive execution time: %f ms\n", elapsedTime);
    #endif

    #ifndef SharedConflict
    hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);
    hipEventRecord(start, 0);
    transposeShared<<<grid, threads>>>(d_out, d_in, width, height);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Transpose Shared (with conflicts) execution time: %f ms\n", elapsedTime);
    #endif

    #ifndef SharedNoConflict
    hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);
    hipEventRecord(start, 0);
    transposeSharedNoBankConflicts<<<grid, threads>>>(d_out, d_in, width, height);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Transpose Shared (no bank conflicts) execution time: %f ms\n", elapsedTime);
    #endif

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_in);
    hipFree(d_out);
    
    delete [] h_in;
    delete [] h_out;

    return 0;
}
